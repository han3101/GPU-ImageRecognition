#include "cuda_image.cuh"
#include <iostream>
#include <cstdlib>

CUDAImageProcessor::CUDAImageProcessor() {}

CUDAImageProcessor::~CUDAImageProcessor() {}


void CUDAImageProcessor::grayscale_avg(Image& image) {

    // Allocate memory buffers
    size_t bytes_n = image.w * image.h * sizeof(uint8_t);
    size_t bytes_i = image.size * sizeof(uint8_t);
    uint8_t* newData = new uint8_t[bytes_n];

    uint8_t *data_d, *result_d;
    hipMalloc(&result_d, bytes_n);
    hipMalloc(&data_d, bytes_i);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID = (bytes_n + THREADS - 1) / THREADS;

    grayscale_avg_cu<<<GRID, THREADS>>>(data_d, result_d, image.channels);
    hipDeviceSynchronize();

    hipMemcpy(newData, result_d, bytes_n, hipMemcpyDeviceToHost);

    delete[] image.data;
    image.data = newData;
    image.size = bytes_n;
    image.channels = 1;

    hipFree(result_d);
    hipFree(data_d);


}

void CUDAImageProcessor::grayscale_lum(Image& image) {

    // Allocate memory buffers
    size_t bytes_n = image.w * image.h * sizeof(uint8_t);
    size_t bytes_i = image.size * sizeof(uint8_t);
    uint8_t* newData = new uint8_t[bytes_n];

    uint8_t *data_d, *result_d;
    hipMalloc(&result_d, bytes_n);
    hipMalloc(&data_d, bytes_i);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID = (bytes_n + THREADS - 1) / THREADS;

    grayscale_lum_cu<<<GRID, THREADS>>>(data_d, result_d, image.channels);
    hipDeviceSynchronize();

    hipMemcpy(newData, result_d, bytes_n, hipMemcpyDeviceToHost);

    delete[] image.data;
    image.data = newData;
    image.size = bytes_n;
    image.channels = 1;

    hipFree(result_d);
    hipFree(data_d);
}
