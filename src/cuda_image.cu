#include "hip/hip_runtime.h"
#include "cuda_image.cuh"
#include <iostream>
#include <cstdlib>
#include <omp.h>

CUDAImageProcessor::CUDAImageProcessor() {}

CUDAImageProcessor::~CUDAImageProcessor() {}


void CUDAImageProcessor::grayscale_avg(Image& image) {

    // Allocate memory buffers
    size_t bytes_n = image.w * image.h * sizeof(uint8_t);
    size_t bytes_i = image.size * sizeof(uint8_t);
    uint8_t* newData = new uint8_t[bytes_n];

    uint8_t *data_d, *result_d;
    hipMalloc(&result_d, bytes_n);
    hipMalloc(&data_d, bytes_i);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID = (bytes_n + THREADS - 1) / THREADS;

    grayscale_avg_cu<<<GRID, THREADS>>>(data_d, result_d, image.channels);
    hipDeviceSynchronize();

    hipMemcpy(newData, result_d, bytes_n, hipMemcpyDeviceToHost);

    delete[] image.data;
    image.data = newData;
    image.size = bytes_n;
    image.channels = 1;

    hipFree(result_d);
    hipFree(data_d);


}

void CUDAImageProcessor::grayscale_lum(Image& image) {

    // Allocate memory buffers
    size_t bytes_n = image.w * image.h * sizeof(uint8_t);
    size_t bytes_i = image.size * sizeof(uint8_t);
    uint8_t* newData = new uint8_t[bytes_n];

    uint8_t *data_d, *result_d;
    hipMalloc(&result_d, bytes_n);
    hipMalloc(&data_d, bytes_i);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID = (bytes_n + THREADS - 1) / THREADS;

    grayscale_lum_cu<<<GRID, THREADS>>>(data_d, result_d, image.channels);
    hipDeviceSynchronize();

    hipMemcpy(newData, result_d, bytes_n, hipMemcpyDeviceToHost);

    delete[] image.data;
    image.data = newData;
    image.size = bytes_n;
    image.channels = 1;

    hipFree(result_d);
    hipFree(data_d);
}


void CUDAImageProcessor::flipX(Image& image) {

    // Allocate memory buffers
    size_t bytes_i = image.size * sizeof(uint8_t);

    uint8_t *data_d;
    hipMalloc(&data_d, bytes_i);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID_X = (image.w + THREADS - 1) / THREADS;
    int GRID_Y = (image.h + THREADS - 1) / THREADS;

    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(GRID_X, GRID_Y);

    flipX_cu<<<grid_dim, block_dim>>>(data_d, image.w, image.h, image.channels);
    hipDeviceSynchronize();

    hipMemcpy(image.data, data_d, bytes_i, hipMemcpyDeviceToHost);

    hipFree(data_d);
}

void CUDAImageProcessor::flipY(Image& image) {

    // Allocate memory buffers
    size_t bytes_i = image.size * sizeof(uint8_t);

    uint8_t *data_d;
    hipMalloc(&data_d, bytes_i);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID_X = (image.w + THREADS - 1) / THREADS;
    int GRID_Y = (image.h + THREADS - 1) / THREADS;

    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(GRID_X, GRID_Y);

    flipY_cu<<<grid_dim, block_dim>>>(data_d, image.w, image.h, image.channels);
    hipDeviceSynchronize();

    hipMemcpy(image.data, data_d, bytes_i, hipMemcpyDeviceToHost);

    hipFree(data_d);
}

void CUDAImageProcessor::flipYvector(Image& image) {

    if (image.channels != 3) {
        std::cout<<"flipYvector only for 3 channel images, using flipY"<<std::endl;
        this->flipY(image);
        return;
    }

    // Allocate memory buffers
    std::vector<uchar3> data_h(image.w * image.h);

    #pragma omp parallel for
    for (int i=0; i < image.w * image.h; i++) {
        data_h[i].x = image.data[i * image.channels];
        data_h[i].y = image.data[i * image.channels + 1];
        data_h[i].z = image.data[i * image.channels + 2];
    }
    

    uchar3* data_d;
    hipMalloc(&data_d, image.w * image.h * sizeof(uchar3));

    hipMemcpy(data_d, data_h.data(), image.w * image.h * sizeof(uchar3), hipMemcpyHostToDevice);

    int GRID_X = (image.w + THREADS - 1) / THREADS;
    int GRID_Y = (image.h + THREADS - 1) / THREADS;

    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(GRID_X, GRID_Y);

    flipYvector_cu<<<grid_dim, block_dim>>>(data_d, image.w, image.h);
    hipDeviceSynchronize();

    hipMemcpy(data_h.data(), data_d, image.w * image.h * sizeof(uchar3), hipMemcpyDeviceToHost);


    #pragma omp parallel for
    for (int i=0; i < image.w * image.h; i++) {
        image.data[i * image.channels] = data_h[i].x;
        image.data[i * image.channels + 1] = data_h[i].y;
        image.data[i * image.channels + 2] = data_h[i].z;
    }

    hipFree(data_d);
}

void CUDAImageProcessor::resizeBilinear(Image& image, int nw, int nh) {

    // Allocate memory buffers
    size_t bytes_i = image.size * sizeof(uint8_t);
    size_t bytes_o = nw * nh * image.channels * sizeof(uint8_t);


    uint8_t *data_d, *output_d;
    hipMalloc(&data_d, bytes_i);
    hipMalloc(&output_d, bytes_o);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID_X = (nw + THREADS - 1) / THREADS;
    int GRID_Y = (nh + THREADS - 1) / THREADS;

    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(GRID_X, GRID_Y);

    float scaleX = (float) (image.w-1) / (nw-1);
    float scaleY = (float) (image.h-1) / (nh-1);

    resize_bilinear_cu<<<grid_dim, block_dim>>>(data_d, output_d, nw, nh, image.w, image.h, image.channels, scaleX, scaleY);
    hipDeviceSynchronize();

    image.size = nw * nh * image.channels;
	uint8_t* newData = new uint8_t[image.size];
    image.w = nw;
	image.h = nh;
	delete[] image.data;
	image.data = newData;
	newData = nullptr;

    hipMemcpy(image.data, output_d, bytes_o, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(output_d);

}

void CUDAImageProcessor::std_convolve_clamp_to_0(Image &image, const Mask::BaseMask *mask) {

    uint32_t MASK_W = mask->getWidth(), MASK_OFFSET_W = mask->getCenterColumn();
    uint32_t MASK_H = mask->getHeight(), MASK_OFFSET_H = mask->getCenterRow();
	const double* ker = mask->getData(); 

    size_t bytes_i = image.size * sizeof(uint8_t);
    size_t bytes_m = MASK_W * MASK_H * sizeof(double);

    uint8_t *data_d, *result_d;
    double *mask_d;
    hipMalloc(&data_d, bytes_i);
    hipMalloc(&result_d, bytes_i);
    hipMalloc(&mask_d, bytes_m);

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);
    hipMemcpy(mask_d, ker, bytes_m, hipMemcpyHostToDevice);

    // if (MASK_H == 3) { 
    //     checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(mask3), ker, bytes_m), "Failed to copy mask3 to constant memory");
    // }
    // if (MASK_H == 5) hipMemcpyToSymbol(HIP_SYMBOL(mask5), ker, bytes_m);

    // hipDeviceSynchronize();

    this->THREADS = 16;

    int GRID_X = (image.w + THREADS - 1) / THREADS;
    int GRID_Y = (image.h + THREADS - 1) / THREADS;

    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(GRID_X, GRID_Y);

    convolve_0_cu<<<grid_dim, block_dim>>>(data_d, result_d, image.w, image.h, image.channels, MASK_H, MASK_OFFSET_H, THREADS, mask_d);
    hipDeviceSynchronize();

    hipMemcpy(image.data, result_d, bytes_i, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(result_d);
}

void CUDAImageProcessor::integralImage(Image& image, std::unique_ptr<u_int32_t[]>& integralImage, std::unique_ptr<u_int32_t[]>& integralImageSquare, std::unique_ptr<u_int32_t[]>& integralImageTilt) {

    if (image.channels > 1) {
        this->grayscale_lum(image);
    }

    // Allocate memory buffers
    size_t bytes_i = image.size * sizeof(uint8_t);
    size_t bytes_o = image.w * image.h * sizeof(u_int32_t);


    uint8_t *data_d;
    u_int32_t *integralImage_d, *integralImageSquare_d, *integralImageTilt_d;
    hipMalloc(&data_d, bytes_i);
    hipMalloc(&integralImage_d, bytes_o);
    if (integralImageSquare) {
        hipMalloc(&integralImageSquare_d, bytes_o);
    }
    if (integralImageTilt) {
        hipMalloc(&integralImageTilt_d, bytes_o);
    }
    

    hipMemcpy(data_d, image.data, bytes_i, hipMemcpyHostToDevice);

    int GRID_X = (image.w + THREADS - 1) / THREADS;
    int GRID_Y = (image.h + THREADS - 1) / THREADS;

    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(GRID_X, GRID_Y);

    integralImage_cu<<<grid_dim, block_dim>>>(data_d, integralImage_d, integralImageSquare_d, integralImageTilt_d, image.w, image.h);
    hipDeviceSynchronize();

    hipMemcpy(integralImage.get(), integralImage_d, bytes_o, hipMemcpyDeviceToHost);
    if (integralImageSquare) {
        hipMemcpy(integralImageSquare.get(), integralImageSquare_d, bytes_o, hipMemcpyDeviceToHost);
    }
    if (integralImageTilt) {
        hipMemcpy(integralImageTilt.get(), integralImageTilt_d, bytes_o, hipMemcpyDeviceToHost);
    }

    hipFree(data_d);
    hipFree(integralImage_d);
    if (integralImageSquare) hipFree(integralImageSquare_d);
    if (integralImageTilt) hipFree(integralImageTilt_d);
}

