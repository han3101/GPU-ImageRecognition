#include "hip/hip_runtime.h"
// #include "cuda_image.cuh"
#include "cuda_image.cuh"

/*
* For optimization use with hipMemcpyToSymbol
* Constant image size might need to be bigger
*/
// __constant__ unsigned char image_data[1024 * 1024];

template <typename T>
__device__ T clamp(T x, T a, T b) {
    return max(a, min(x, b));
}

__global__ void grayscale_avg_cu(const unsigned char *data, unsigned char *output, int channels) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int pixelIndex = id * channels;

    unsigned char r = data[pixelIndex];
    unsigned char g = data[pixelIndex + 1];
    unsigned char b = data[pixelIndex + 2];

    unsigned char gray = (r + g + b) / 3;

    output[id] = gray;

} 

__global__ void grayscale_lum_cu(const unsigned char *data, unsigned char *output, int channels) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int pixelIndex = id * channels;

    unsigned char r = data[pixelIndex];
    unsigned char g = data[pixelIndex + 1];
    unsigned char b = data[pixelIndex + 2];

    unsigned char gray = (r * 0.299 + g * 0.5870 + 0.1140 * b);

    output[id] = gray;
} 


__global__ void flipX_cu(unsigned char *data, int w, int h, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w / 2 && y < h) {
        int left = (x + y*w) * channels;
        int right = ((w - 1- x) + y*w) * channels;

        for (int c=0; c < channels; ++c) {
            unsigned char tmp = data[left + c];
            data[left + c] = data[right + c];
            data[right + c] = tmp;
        }
    }
}

__global__ void flipY_cu(unsigned char *data, int w, int h, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h / 2) {
        int left = (x + y*w) * channels;
        int right = (x + (h - y + 1)*w) * channels;

        for (int c=0; c < channels; ++c) {
            unsigned char tmp = data[left + c];
            data[left + c] = data[right + c];
            data[right + c] = tmp;
        }
    }
}


__global__ void flipYvector_cu(uchar3 *data, int w, int h) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h / 2) {
        int left = x + y * w;
        int right = (x + (h - y + 1) * w);

        uchar3 tmp = data[left];
        data[left] = data[right];
        data[right] = tmp;
    }
}

__global__ void resize_bilinear_cu(unsigned char *data, unsigned char *output, int nw, int nh, int w, int h, int channels, float scaleX, float scaleY) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < nh && col < nw) {
        float float_pos_x, float_pos_y, offset_x, offset_y;
        ushort low_x, low_y, high_x, high_y; 

        float_pos_x = col * scaleX;
        float_pos_y = row * scaleY;

        low_x = (ushort)floor(float_pos_x);
        low_y = (ushort)floor(float_pos_y);
        high_x = low_x + 1;
        high_y = low_y + 1;

        if (high_x >= w) {
            high_x = low_x;
        }
        if (high_y >= h) {
            high_y = low_y;
        }

        offset_x = float_pos_x - low_x;
        offset_y = float_pos_y - low_y;

        for (int c = 0; c < channels; ++c) {
            
            float value = (1-offset_x) * (1-offset_y) * data[(low_x + low_y * w) * channels + c] +
                        offset_x * (1-offset_y) * data[(high_x + low_y * w) * channels + c] +
                        (1-offset_x) * offset_y * data[(low_x + high_y * w) * channels + c] +
                        offset_x * offset_y * data[(high_x + high_y * w) * channels + c];

            output[(col + row * nw) * channels + c] = clamp(value, 0.0f, 255.0f);
        }


    }
}