#include "hip/hip_runtime.h"
// #include "cuda_image.cuh"
#include "cuda_image.cuh"

/*
* For optimization use with hipMemcpyToSymbol
* Constant image size might need to be bigger
*/
// __constant__ unsigned char image_data[1024 * 1024];

template <typename T>
__device__ T clamp(T x, T a, T b) {
    return max(a, min(x, b));
}

__global__ void grayscale_avg_cu(const unsigned char *data, unsigned char *output, int channels) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int pixelIndex = id * channels;

    unsigned char r = data[pixelIndex];
    unsigned char g = data[pixelIndex + 1];
    unsigned char b = data[pixelIndex + 2];

    unsigned char gray = (r + g + b) / 3;

    output[id] = gray;

} 

__global__ void grayscale_lum_cu(const unsigned char *data, unsigned char *output, int channels) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int pixelIndex = id * channels;

    unsigned char r = data[pixelIndex];
    unsigned char g = data[pixelIndex + 1];
    unsigned char b = data[pixelIndex + 2];

    unsigned char gray = (r * 0.299 + g * 0.5870 + 0.1140 * b);

    output[id] = gray;
} 


__global__ void flipX_cu(unsigned char *data, int w, int h, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w / 2 && y < h) {
        int left = (x + y*w) * channels;
        int right = ((w - 1- x) + y*w) * channels;

        for (int c=0; c < channels; ++c) {
            unsigned char tmp = data[left + c];
            data[left + c] = data[right + c];
            data[right + c] = tmp;
        }
    }
}

__global__ void flipY_cu(unsigned char *data, int w, int h, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h / 2) {
        int left = (x + y*w) * channels;
        int right = (x + (h - y + 1)*w) * channels;

        for (int c=0; c < channels; ++c) {
            unsigned char tmp = data[left + c];
            data[left + c] = data[right + c];
            data[right + c] = tmp;
        }
    }
}


__global__ void flipYvector_cu(uchar3 *data, int w, int h) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h / 2) {
        int left = x + y * w;
        int right = (x + (h - y + 1) * w);

        uchar3 tmp = data[left];
        data[left] = data[right];
        data[right] = tmp;
    }
}

__global__ void resize_bilinear_cu(unsigned char *data, unsigned char *output, int nw, int nh, int w, int h, int channels, float scaleX, float scaleY) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < nh && col < nw) {
        float float_pos_x, float_pos_y, offset_x, offset_y;
        ushort low_x, low_y, high_x, high_y; 

        float_pos_x = col * scaleX;
        float_pos_y = row * scaleY;

        low_x = (ushort)floor(float_pos_x);
        low_y = (ushort)floor(float_pos_y);
        high_x = low_x + 1;
        high_y = low_y + 1;

        if (high_x >= w) {
            high_x = low_x;
        }
        if (high_y >= h) {
            high_y = low_y;
        }

        offset_x = float_pos_x - low_x;
        offset_y = float_pos_y - low_y;

        for (int c = 0; c < channels; ++c) {
            
            float value = (1-offset_x) * (1-offset_y) * data[(low_x + low_y * w) * channels + c] +
                        offset_x * (1-offset_y) * data[(high_x + low_y * w) * channels + c] +
                        (1-offset_x) * offset_y * data[(low_x + high_y * w) * channels + c] +
                        offset_x * offset_y * data[(high_x + high_y * w) * channels + c];

            output[(col + row * nw) * channels + c] = clamp(value, 0.0f, 255.0f);
        }


    }
}


__global__ void integralImage_cu(unsigned char *data, u_int32_t *integralImage, u_int32_t *integralImageSquare, u_int32_t *integralImageTilt, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= height || col >= width) {
        return;
    }

    // Load initial data into integral images
    integralImage[row * width + col] = static_cast<uint32_t>(data[row * width + col]);
    if (integralImageSquare) {
			integralImageSquare[row * width + col] = static_cast<uint32_t>(data[row * width + col] * data[row * width + col]);
		}
    if (integralImageTilt) {
        integralImageTilt[row * width + col] = static_cast<uint32_t>(data[row * width + col]);
    }

    __syncthreads();
    
    // Row prefix
    if (col == 0) {
        for (int i = 1; i < width; i++) {
            integralImage[row * width + i] += integralImage[row * width + (i-1)];
            if (integralImageSquare) {
                integralImageSquare[row * width + i] += integralImageSquare[row * width + (i-1)];
            }
            // RSAT
            if (integralImageTilt && row > 0) {
                integralImageTilt[row * width + i] += integralImageTilt[(row-1) * width + (i-1)];
            }
        }
    }
    

    __syncthreads();

    // Col prefix sum
    if (row == 0) {
        for (int j=1; j < height; j++) {
            integralImage[j * width + col] += integralImage[(j-1) * width + col];
            if (integralImageSquare) {
                integralImageSquare[j * width + col] += integralImageSquare[(j-1) * width + col];
            }

            if (integralImageTilt) {
				integralImageTilt[j * width + col] += integralImageTilt[(j - 1) * width + col];
				if (col > 0) {
					integralImageTilt[j * width + col] += integralImageTilt[(j - 1) * width + (col - 1)];
				}
				if (col < width - 1) {
					integralImageTilt[j * width + col] += integralImageTilt[(j - 1) * width + (col + 1)];
				}
				if (j > 1) {
					integralImageTilt[j * width + col] -= integralImageTilt[(j - 2) * width + col];
				}
			}
        }
    }

}